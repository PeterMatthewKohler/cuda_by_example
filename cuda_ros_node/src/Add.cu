#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>



namespace cuda_ros_node {
__global__ void add(int* a, int* b, int* c, int* arrSize)
{
    int tid = blockIdx.x; // Perform operation on this index
    if(tid < *arrSize)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void cudaAdd(int* a, int* b, int* c, int arrSize)
{
    // Pointers to the memory we will allocate on the device
    int *dev_a, *dev_b, *dev_c, *dev_ArrSize;

    // Allocate the memory on the device
    hipMalloc((void **)&dev_a, arrSize * sizeof(int));
    hipMalloc((void **)&dev_b, arrSize * sizeof(int));
    hipMalloc((void **)&dev_c, arrSize * sizeof(int));
    hipMalloc((void **)&dev_ArrSize, sizeof(int));
    // Copy inputs to device
    hipMemcpy(dev_a, a, arrSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arrSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ArrSize, &arrSize, sizeof(int), hipMemcpyHostToDevice);
    // Launch add() kernel
    add<<<arrSize,1>>>(dev_a, dev_b, dev_c, dev_ArrSize);
    // Copy results back to the host
    hipMemcpy(c, dev_c, arrSize * sizeof(int), hipMemcpyDeviceToHost);
    // Cleanup allocated memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return;
}

void cudaPrintDeviceProperties()
{
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount(&count);
    printf("Number of CUDA devices: %d\n", count);
    for(int i=0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("Device %d\n", i);
        printf("  Name: %s\n", prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total global memory: %lu\n", prop.totalGlobalMem);
        printf("  Shared memory per block: %lu\n", prop.sharedMemPerBlock);
        printf("  Registers per block: %d\n", prop.regsPerBlock);
        printf("  Warp size: %d\n", prop.warpSize);
        printf("  Threads in warp: %d\n", prop.maxThreadsPerBlock);
        printf("  Max thread dimensions: %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max grid dimensions: %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
}
} // namespace cuda_ros_node